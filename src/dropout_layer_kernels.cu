#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "rocrand/rocrand.h"
#include "rocblas.h"

#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

void forward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if (!net.train) return;
    int size = layer.inputs*layer.batch;
    hip_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    hip_push_array(layer.rand_gpu, layer.rand, size);
    */

    hipLaunchKernelGGL((yoloswag420blazeit360noscope), dim3(hip_gridsize(size)), dim3(BLOCK), 0, 0, net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if(!net.delta_gpu) return;
    int size = layer.inputs*layer.batch;

    hipLaunchKernelGGL((yoloswag420blazeit360noscope), dim3(hip_gridsize(size)), dim3(BLOCK), 0, 0, net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}
